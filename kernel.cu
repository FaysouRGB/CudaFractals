#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <opencv2/opencv.hpp>
#include "Settings.h"
#include "Complex.hpp"
#include <cmath>

using namespace cv;
using namespace std;

void AskSettings(Settings* settings)
{
	cout << "Enter image height: ";
	cin >> settings->image_height;

	cout << "Enter image width: ";
	cin >> settings->image_width;

	cout << "Enter max iterations: ";
	cin >> settings->max_iterations;

	cout << "Settings are: " << settings->image_height << "x" << settings->image_width << " - Max iterations: " << settings->max_iterations << endl;
}

int main()
{
	// Settings
	auto settings = Settings();
	AskSettings(&settings);

	// Create the image
	Mat image = Mat(static_cast<int>(settings.image_height), static_cast<int>(settings.image_width), CV_8UC3, Scalar(0, 0, 0));

	// Mandelbrot
	size_t y0;
	size_t x0;
	for (size_t y = 0; y < settings.image_height; y++)
	{
		for (size_t x = 0; x < settings.image_width; x++)
		{
			// Save indices
			y0 = y;
			x0 = x;
			lerp
				double X = static_cast<double>(x0);
			double Y = static_cast<double>(y0);
			size_t iteration = 0;

			while (x * x + y * y <= 4 && iteration < settings.max_iterations)
			{
				auto xtemp = x * x - y * y + x0;
				y = 2 * x * y + y0;

				x = xtemp;

				iteration += 1;
			}

			if (iteration != settings.max_iterations)
			{
				Vec3b& color = image.at<Vec3b>(y0, x0);
				color[0] = 255;
			}

			// Put them back
			x = x0;
			y = y0;
		}
	}

	// Save the mandelbrot image
	imwrite("mandelbrot.png", image);

	return 0;
}